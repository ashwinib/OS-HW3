#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stddef.h>
#include<search.h>
#include<hip/device_functions.h>
#define MAX_FILE_SIZE 200
#define MAX_HASH_ENTRIES 200
#define M 10

__global__ void getWordCounts(char *fileArray,int *countArray,int *fileSize,char *wordhashtable, int *nextPtr, int *lock){
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  int ind,word_started =0 ,count =0;
  int found;
  int hashvalue;
  char *ptr,*wptr,*temp;
  ptr = &fileArray[i*MAX_FILE_SIZE];
  int  tempi=0,tempi2;

  for(ind =0;ind<fileSize[i];ind++){
    if(ptr[ind]>64&&ptr[ind]<91) ptr[ind]+=32;
    if(ptr[ind]!=' '&&ptr[ind]!='.'&&ptr[ind]!='!')
      if(word_started!=1) {
	word_started = 1;
	hashvalue = ptr[ind];//>64&&ptr[ind]<91) ? ptr[ind]+32:ptr[ind];//temp addition else do only assignemnt
	wptr = &ptr[ind];
      }
      else{//Middle of the word
	hashvalue+= ptr[ind];//>64&&ptr[ind]<91) ? ptr[ind]+32:ptr[ind];
      }
    if(word_started)
      if(ptr[ind]==' '||ptr[ind]=='.'||ptr[ind]=='!'){
        word_started = 0;
	hashvalue = hashvalue % M;// 10 here is hashtable size M

	/*Check Location*/	
	//lock -hashvalue
	while(!atomicCAS(&lock[hashvalue],0,1));
	if(wordhashtable[hashvalue*20]=='\0'){//Not found in Hash
	  temp = &wordhashtable[hashvalue*20];
	  tempi =0;
	  while(&wptr[tempi]!=&ptr[ind])//Entering in hash table
		{temp[tempi]= wptr[tempi];
		tempi++;}
	//unlock -hash value
	  atomicCAS(&lock[hashvalue],1,0);
	  //fn-atomicAdd(&countArray[hashvalue],1);//count

  countArray[hashvalue] = hashvalue; 
	}
	else{//Collision detection
	  tempi =hashvalue;found = -1;
	
	 /*Check word*/
	  while(nextPtr[tempi]!=-1||found==-1){
	    tempi2 = 0;
	    found =1;
	    temp = &wordhashtable[tempi*20];
	    while(&wptr[tempi2]!=&ptr[ind]){
	      if(temp[tempi2]!=wptr[tempi2]) {found =0;break;}
	      tempi2++;
	    }
	    if(temp[tempi2]!='\0') found =0;
	    //unlock - tempi
	    atomicCAS(&lock[tempi],1,0);
	    if(found) break;
	    if(nextPtr[tempi]!=-1){
	       	tempi = nextPtr[tempi];      
		//lock - tempi
		while(!atomicCAS(&lock[tempi],0,1));
	    }
	  }

	  if(found){
	    atomicAdd(&countArray[tempi],1);
		countArray[tempi]=hashvalue;}//DEBUG
	  else{//Collision but record not found
	    tempi2 =0;
	    //lock - M+tempi2
	    while(!atomicCAS(&lock[M+tempi2],0,1));
	    while(wordhashtable[(M+tempi2)*20]!='\0' && tempi2<MAX_HASH_ENTRIES) tempi2++;//10 = M; tempi2 holds location in hast tab;e
	    if(tempi2 < MAX_HASH_ENTRIES){
	    	nextPtr[tempi] = tempi2+M;tempi=0;//tempi holds the location where last hash was found
	        temp = &wordhashtable[(M+tempi2)*20];
		while(&wptr[tempi]!=&ptr[ind]) //Entering in hash table
			{temp[tempi]= wptr[tempi]; 
			tempi++;}
		//unlock - M+tempi2
	        atomicCAS(&lock[M+tempi2],1,0);

  countArray[tempi2+M] = hashvalue; 
		//fn-atomicAdd(&countArray[tempi2+M],1);
	    }//count*/
	    //tryunlock = M+tempi2
	    atomicCAS(&lock[M+tempi2],1,0);
	  }

	}
	//atomicAdd(&countArray[hashvalue],1);
	//atomicExch(&countArray[hashvalue],hashvalue);
	count++;
      }
  }
  //countArray[i] = hashvalue; 
}

int main(int argc,char **argv){
  char *filename=NULL;//Limiting no if files
  char *fileArray;
  char *dfileArray;
  int *countArray;
  int *dcountArray;
  int *fileSize;
  int *dfileSize;
  char *hashtable; 
  char *dhashtable; 
  int *nextPtr;
  int *dnextPtr;
  int *dlock;
  int noOfFiles=0;
  FILE *fp;

  char *temp;int itemp=0;
  filename =(char*) malloc (10*sizeof(char));
  fileArray=(char*) malloc(10*MAX_FILE_SIZE*sizeof(char));
  countArray =(int*) malloc (MAX_HASH_ENTRIES*sizeof(int));//corresponding counts of words
  fileSize =(int*) malloc (10*sizeof(int));
  hashtable=(char*) malloc(20*MAX_HASH_ENTRIES*sizeof(char));
  nextPtr = (int*) malloc (MAX_HASH_ENTRIES*sizeof(int));

  hipMalloc((void**)&dfileArray,10*MAX_FILE_SIZE*sizeof(char));
  hipMalloc((void**)&dcountArray,MAX_HASH_ENTRIES*sizeof(int));//corresponding counts of words
  hipMalloc((void**)&dfileSize,10*sizeof(int));
  hipMalloc((void**)&dhashtable,20*MAX_HASH_ENTRIES*sizeof(char));//20-max word size 500-max words
  hipMalloc((void**)&dnextPtr,MAX_HASH_ENTRIES*sizeof(int));//corresponding counts of words
  hipMalloc((void**)&dlock,MAX_HASH_ENTRIES*sizeof(int));//corresponding counts of words

  hipMemset(dcountArray,0,MAX_HASH_ENTRIES*sizeof(int));
  hipMemset(dhashtable,'\0',20*MAX_HASH_ENTRIES*sizeof(char));
  hipMemset(dnextPtr,-1,MAX_HASH_ENTRIES*sizeof(int));
  hipMemset(dlock,0,MAX_HASH_ENTRIES*sizeof(int));
  
  while(scanf("%s",filename)!=EOF){
    printf("\nAttempting to open %s",filename);
    fp = fopen(filename,"r");
    if(fp == NULL) {
	        perror("failed to open sample.txt");
        	exit(0) ;//EXIT_FAILURE;
    }
    fread(&fileArray[noOfFiles*200],MAX_FILE_SIZE*sizeof(char),1,fp);
    fileSize[noOfFiles]=ftell(fp);
    fclose(fp);fp = NULL;
    noOfFiles++;
  }

  temp = fileArray;
  while(itemp<noOfFiles){
    printf("%s\n",temp);itemp++;
    temp+=200;
  }
  hipMemcpy(dfileArray,fileArray,10*MAX_FILE_SIZE*sizeof(char),hipMemcpyHostToDevice);
  hipMemcpy(dfileSize,fileSize,10*sizeof(int),hipMemcpyHostToDevice);
  getWordCounts<<<1,noOfFiles>>>(dfileArray,dcountArray,dfileSize,dhashtable,dnextPtr, dlock);
  hipDeviceSynchronize();
  hipMemcpy(countArray,dcountArray,200*sizeof(int),hipMemcpyDeviceToHost);
  hipMemcpy(hashtable,dhashtable,20*200*sizeof(char),hipMemcpyDeviceToHost);

  itemp=0;
  printf("\nNo Of Words : \n");
  while(itemp<200){
//    printf("\t%d",countArray[itemp]);itemp++;
    if(hashtable[itemp*20]!='\0')
      printf("%s:[%d]\n",&hashtable[itemp*20],countArray[itemp]);
    itemp++;
  }
  hipFree(dfileArray);
  hipFree(dcountArray);
  hipFree(dhashtable);
  free(fileArray);
  free(countArray);
  free(hashtable);
}

#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stddef.h>
#include<search.h>
#include<hip/device_functions.h>
#define MAX_FILE_SIZE 200*sizeof(char)

__global__ void getWordCounts(char *fileArray,int *countArray,int *fileSize,char *wordhashtable){
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  int ind,word_started =0 ,count =0;
  int hashvalue;
  char *ptr,*wptr,*temp;
  ptr = &fileArray[i*200];int  tempi=0;
  for(ind =0;ind<fileSize[i];ind++){
    if(ptr[ind]!=' '&&ptr[ind]!='.'&&ptr[ind]!='!')
      if(word_started!=1) {
	word_started = 1;
	hashvalue = (ptr[ind]>64&&ptr[ind]<91) ? ptr[ind]+32:ptr[ind];//temp addition else do only assignemnt
	wptr = &ptr[ind];
      }
      else{//Middle of the word
	hashvalue+= (ptr[ind]>64&&ptr[ind]<91) ? ptr[ind]+32:ptr[ind];
      }
    if(word_started)
      if(ptr[ind]==' '||ptr[ind]=='.'||ptr[ind]=='!'){
        word_started = 0;
	hashvalue = hashvalue % 100;
	if(wordhashtable[hashvalue*20]=='\0'){
	temp = &wordhashtable[hashvalue*20];tempi =0;
	while(&wptr[tempi]!=&ptr[ind]){temp[tempi]=wptr[tempi];tempi++;}///TODO: MAKE ATOMIC!!!
	}
	atomicAdd(&countArray[hashvalue],1);
	//atomicExch(&countArray[hashvalue],hashvalue);
	count++;
	//break;//temmporary for testing
      }
  }
  //countArray[i] = hashvalue; 
}

int main(int argc,char **argv){
  char *filename=NULL;//Limiting no if files
  char *fileArray;
  char *dfileArray;
  int *countArray;
  int *dcountArray;
  int *fileSize;
  int *dfileSize;
  char *hashtable; 
  char *dhashtable; 
  int noOfFiles=0;
  FILE *fp;

  char *temp;int itemp=0;
  filename =(char*) malloc (10*sizeof(char));
  fileArray=(char*) malloc(10*MAX_FILE_SIZE);
  countArray =(int*) malloc (200*sizeof(int));//corresponding counts of words
  fileSize =(int*) malloc (10*sizeof(int));
  hashtable=(char*) malloc(20*200*sizeof(char));
  hipMalloc((void**)&dfileArray,10*MAX_FILE_SIZE);
  hipMalloc((void**)&dcountArray,200*sizeof(int));//corresponding counts of words
  hipMalloc((void**)&dfileSize,10*sizeof(int));
  hipMalloc((void**)&dhashtable,20*200*sizeof(char));//20-max word size 500-max words
  hipMemset(dcountArray,0,200*sizeof(int));
  hipMemset(dhashtable,'\0',20*200*sizeof(char));
  
  while(scanf("%s",filename)!=EOF){
    printf("\nAttempting to open %s",filename);
    fp = fopen(filename,"r");
    if(fp == NULL) {
	        perror("failed to open sample.txt");
        	exit(0) ;//EXIT_FAILURE;
    }
    fread(&fileArray[noOfFiles*200],MAX_FILE_SIZE,1,fp);
    fileSize[noOfFiles]=ftell(fp);
    fclose(fp);fp = NULL;
    noOfFiles++;
  }

  temp = fileArray;
  while(itemp<noOfFiles){
    printf("%s\n",temp);itemp++;
    temp+=200;
  }
  hipMemcpy(dfileArray,fileArray,10*MAX_FILE_SIZE,hipMemcpyHostToDevice);
  hipMemcpy(dfileSize,fileSize,10*sizeof(int),hipMemcpyHostToDevice);
  getWordCounts<<<1,noOfFiles>>>(dfileArray,dcountArray,dfileSize,dhashtable);
  hipDeviceSynchronize();
  hipMemcpy(countArray,dcountArray,200*sizeof(int),hipMemcpyDeviceToHost);
  hipMemcpy(hashtable,dhashtable,20*200*sizeof(char),hipMemcpyDeviceToHost);

  itemp=0;
  printf("\nNo Of Words : \n");
  while(itemp<200){
//    printf("\t%d",countArray[itemp]);itemp++;
    if(hashtable[itemp*20]!='\0')
      printf("%s:[%d]\n",&hashtable[itemp*20],countArray[itemp]);
    itemp++;
  }
  hipFree(dfileArray);
  hipFree(dcountArray);
  hipFree(dhashtable);
  free(fileArray);
  free(countArray);
  free(hashtable);
}

#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stddef.h>
#include<search.h>
#define MAX_FILE_SIZE 200*sizeof(char)

__global__ void getWordCounts(char *fileArray,int *countArray,int *fileSize,int *hashtable){
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  int ind,word_started =0 ,count =0;
  int hashvalue;
  char *ptr,*wptr;
  ptr = &fileArray[i*200];int  tempi=0;
  for(ind =0;ind<fileSize[i];ind++){
    if(ptr[ind]!=' '&&ptr[ind]!='.'&&ptr[ind]!='!')
      if(word_started!=1) {word_started = 1;hashvalue = (ptr[ind]>64&&ptr[ind]<91) ? ptr[ind]+32:ptr[ind];}
      else{
	hashvalue+= (ptr[ind]>64&&ptr[ind]<91) ? ptr[ind]+32:ptr[ind];
      }
    if(word_started)
      if(ptr[ind]==' '||ptr[ind]=='.'||ptr[ind]=='!'){
        word_started = 0;
	hashvalue = hashvalue % 100;
	count++;
	break;//temmporary for testing
      }
  }
  countArray[i] = hashvalue; 
}

int main(int argc,char **argv){
  char *filename=NULL;//Limiting no if files
  char *fileArray;
  char *dfileArray;
  int *countArray;
  int *dcountArray;
  int *fileSize;
  int *dfileSize;
  
  int noOfFiles=0;
  FILE *fp;

  char *temp;int itemp=0;
  filename =(char*) malloc (10*sizeof(char));
  fileArray=(char*) malloc(10*MAX_FILE_SIZE);
  countArray =(int*) malloc (10*sizeof(int));
  fileSize =(int*) malloc (10*sizeof(int));
  hipMalloc((void**)&dfileArray,10*MAX_FILE_SIZE);
  hipMalloc((void**)&dcountArray,10*sizeof(int));
  hipMalloc((void**)&dfileSize,10*sizeof(int));
  hipMemset(dcountArray,0,10*sizeof(int));
  
  while(scanf("%s",filename)!=EOF){
    printf("\nAttempting to open %s",filename);
    fp = fopen(filename,"r");
    if(fp == NULL) {
	        perror("failed to open sample.txt");
        	exit(0) ;//EXIT_FAILURE;
    }
    fread(&fileArray[noOfFiles*200],MAX_FILE_SIZE,1,fp);
    fileSize[noOfFiles]=ftell(fp);
    fclose(fp);fp = NULL;
    noOfFiles++;
  }

  temp = fileArray;
  while(itemp<noOfFiles){
    printf("%s\n",temp);itemp++;
    temp+=200;
  }
  hipMemcpy(dfileArray,fileArray,10*MAX_FILE_SIZE,hipMemcpyHostToDevice);
  hipMemcpy(dfileSize,fileSize,10*sizeof(int),hipMemcpyHostToDevice);
  getWordCounts<<<1,noOfFiles>>>(dfileArray,dcountArray,dfileSize);
  hipDeviceSynchronize();
  hipMemcpy(countArray,dcountArray,10*sizeof(int),hipMemcpyDeviceToHost);

  itemp=0;
  printf("\nNo Of Words : ");
  while(itemp<noOfFiles){
    printf("\t%d",countArray[itemp]);itemp++;
  }
  hipFree(dfileArray);
  hipFree(dcountArray);
  free(fileArray);
  free(countArray);
}
